#include "LightMapper.cuh"
#include <vector>

void InitMeshBuffers(Mesh* mesh)
{
	g_VertCount = mesh->GetVertices().size();
	g_IndicesCount = mesh->GetIndices().size();

	// Alocate vertex in buffer
	hipFree(g_pVerteInBuffer);
	hipMalloc(&g_pVerteInBuffer, g_VertCount * sizeof(Vertex));
	hipMemcpy(g_pVerteInBuffer, mesh->GetVertices().data(), sizeof(Vertex) * g_VertCount, hipMemcpyHostToDevice);

	// Allocate indices in buffer
	hipFree(g_pIndexBuffer);
	hipMalloc(&g_pIndexBuffer, g_IndicesCount * sizeof(unsigned int));
	hipMemcpy(g_pIndexBuffer, mesh->GetIndices().data(), sizeof(unsigned int) * g_IndicesCount, hipMemcpyHostToDevice);

	checkCUDAError("InitMeshBuffers");
}

void GenerateLightMap(const std::vector<Mesh*>& pMeshes)
{
	for (Mesh* pMesh : pMeshes)
	{
		// Init Mesh buffers
		InitMeshBuffers(pMesh);

		// Render Directional light
		


		// Render pointlight lightmaps


		// Assamble lightmaps
	}
}
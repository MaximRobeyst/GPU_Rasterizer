#include "hip/hip_runtime.h"
#include "gpu.h"
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <vector>
#include <math.h>

#include "Triangle.h"
#include "hip/hip_vector_types.h"

uint32_t* gpuAlloc(void) {
	uint32_t* gpu_mem;

	hipError_t err = hipMalloc(&gpu_mem, SCREEN_SIZE * 4);
	if (err != hipSuccess) return NULL;

	return gpu_mem;
};

void gpuFree(void* gpu_mem) {
	hipFree(gpu_mem);
}

int gpuBlit(void* src, void* dst) {
	hipError_t err = hipMemcpy(dst, src, SCREEN_SIZE * 4, hipMemcpyDeviceToHost);
	if (err != hipSuccess) return 1;
	return 0;
}

__host__
__device__
uint32_t getPixColor(int x, int y) {
	Triangle g_Triangle{ float3{0.0f, 0.5f, -1.0f}, float3{-0.5, -0.5f, -1.0f}, float3{0.5f, -0.5f, -1.0f} };

	float4 color{0.0f, 0.0f, 0.0f, 0.0f};

	float3 pixel{ static_cast<float>(x), static_cast<float>(y), 0.0f };

	// Triangle
	float3 a = make_float3(g_Triangle.m_ViewSpaceVertices[1]) - make_float3(g_Triangle.m_ViewSpaceVertices[0]);
	float3 b = make_float3(g_Triangle.m_ViewSpaceVertices[2]) - make_float3(g_Triangle.m_ViewSpaceVertices[0]);

	auto normal = cross(a, b);

	if (dot(normal, pixel) == 0)
		color = float4{ 1.0f, 1.0f, 1.0f, 1.0f };

	return (uint8_t)(color.x * 255) | ((uint8_t)(color.y * 255) << 8) | ((uint8_t)(color.z * 255) << 16) | ((uint8_t)(color.w * 255) << 24);
}

__global__ void my_kernel(uint32_t* buf) {
	const int xPix = blockDim.x * blockIdx.x + threadIdx.x;
	const int yPix = blockDim.y * blockIdx.y + threadIdx.y;

	unsigned int pos = SCREEN_WIDTH * yPix + xPix;

	buf[pos] = getPixColor(xPix, yPix);
}

void gpuRender(uint32_t* buf) {
	const dim3 blocksPerGrid(H_TILES, V_TILES);
	const dim3 threadsPerBlock(TILE_WIDTH, TILE_HEIGHT);
	my_kernel <<<blocksPerGrid, threadsPerBlock>>> (buf);
}